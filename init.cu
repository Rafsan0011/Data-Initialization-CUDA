#include <thrust/device_vector.h>
#include <thrust/tabulate.h>
#include <iostream>

struct Fragment
{
   int index[3];
   Fragment() = default;
};

struct functor
{
    __device__ __host__
    Fragment operator() (const int &i) const { 
        Fragment f; 
        f.index[0] = i; f.index[1] = i+1; f.index[2] = i+2; 
        return f;
    }
};


int main()
{
    const int N = 10;
    thrust::device_vector<Fragment> dvFragment(N);
    thrust::tabulate(dvFragment.begin(), dvFragment.end(), functor());

    for(auto p : dvFragment) {
        Fragment f = p;
        std::cout << f.index[0] << " " << f.index[1] << " " << f.index[2] << std::endl;
    }

    return 0;
}    